
#include "hip/hip_runtime.h"

#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define W 25
#define H 25

__global__ void kernel(int* a, size_t pitch)
{
	int x = threadIdx.x;
	int y = threadIdx.y;
	int *row_a = (int*)((char*)a + y * pitch);
	// Clear to zero
	row_a[x] = 0;
}

int main()
{
	int **a;
	int *dev_a;
	size_t pitch;
	dim3 threads(W, H);
	// allocate memory for array a
	a = (int**)malloc(H * sizeof(int*));
	
	for (int i = 0; i < H; i++)
	{
		a[i] = (int*)malloc(W * sizeof(int));
	}
	// initialize array a
	for (int i = 0; i < H; i++)
	{
		for (int j = 0; j < W; j++)
		{
			a[i][j] = 1;
		}
	}
	for (int i = 0; i < H; i++)
	{
		for (int j = 0; j < W; j++)
		{
			printf("%d ", a[i][j]);
		}
		printf("\n");
	}
	hipMallocPitch((void**)&dev_a, &pitch, W * sizeof(int), H);	
	hipMemcpy2D(dev_a, pitch, a, W * sizeof(int), W * sizeof(int), H, hipMemcpyHostToDevice);
	kernel<<<1, threads>>>(dev_a, pitch);
	hipMemcpy2D(a, W * sizeof(int), dev_a, pitch, W * sizeof(int), H, hipMemcpyDeviceToHost);

	for (int i = 0; i < H; i++)
	{
		for (int j = 0; j < W; j++)
		{
			printf("%d ", a[i][j]);
		}
		printf("\n");
	}
	printf("\n");
	return 0;
}
